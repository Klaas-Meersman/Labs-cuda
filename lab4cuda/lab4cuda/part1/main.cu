/*
 * Code snippet for importing / exporting image data.
 *
 * To convert an image to a pixel map, run `convert <name>.<extension> <name>.ppm
 *
 */
#include <cstdint>  // Data types
#include <iostream> // File operations
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include <iostream>
#include <vector>
#include <random>
#include <ctime>
#include <algorithm>
#include <math.h>

// #define M 512       // Lenna width
// #define N 512       // Lenna height
#define M 960     // VR width
#define N 1280    // VR height
#define C 3       // Colors
#define OFFSET 16 // Header length

uint8_t *get_image_array(void){
    /*
     * Get the data of an (RGB) image as a 1D array.
     *
     * Returns: Flattened image array.
     *
     * Noets:
     *  - Images data is flattened per color, column, row.
     *  - The first 3 data elements are the RGB components
     *  - The first 3*M data elements represent the firts row of the image
     *  - For example, r_{0,0}, g_{0,0}, b_{0,0}, ..., b_{0,M}, r_{1,0}, ..., b_{b,M}, ..., b_{N,M}
     *
     */
    // Try opening the file
    FILE *imageFile;
    imageFile = fopen("./anna.ppm", "rb");
    if (imageFile == NULL)
    {
        perror("ERROR: Cannot open output file");
        exit(EXIT_FAILURE);
    }

    // Initialize empty image array
    uint8_t *image_array = (uint8_t *)malloc(M * N * C * sizeof(uint8_t) + OFFSET);

    // Read the image
    fread(image_array, sizeof(uint8_t), M * N * C * sizeof(uint8_t) + OFFSET, imageFile);

    // Close the file
    fclose(imageFile);

    // Move the starting pointer and return the flattened image array
    return image_array + OFFSET;
}

void save_image_array(uint8_t *image_array){
    /*
     * Save the data of an (RGB) image as a pixel map.
     *
     * Parameters:
     *  - param1: The data of an (RGB) image as a 1D array
     *
     */
    // Try opening the file
    FILE *imageFile;
    imageFile = fopen("./output_image.ppm", "wb");
    if (imageFile == NULL)
    {
        perror("ERROR: Cannot open output file");
        exit(EXIT_FAILURE);
    }

    // Configure the file
    fprintf(imageFile, "P6\n");          // P6 filetype
    fprintf(imageFile, "%d %d\n", M, N); // dimensions
    fprintf(imageFile, "255\n");         // Max pixel

    // Write the image
    fwrite(image_array, 1, M * N * C, imageFile);

    // Close the file
    fclose(imageFile);
}

struct info{
    float kernelTime;
    float totalTime;
};

__global__ void REDfilterGPU_Uncoalesced(uint8_t *original, uint8_t *inverted, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size){
        if (i % 3 == 0){
            inverted[i] = 255 - original[i];
        }
    }
}


info REDfilterGPUrun_Uncoalesced(){
    // Read the image
    uint8_t *image_array = get_image_array();
    // Allocate host output
    uint8_t *new_image_array = (uint8_t *)malloc(M * N * C);

    // Set up grid and block dimensions
    int blockSize = 256;
    int numBlocks = (M * N * C + blockSize - 1) / blockSize;

    // Allocate device memory
    uint8_t *d_original, *d_inverted;
    hipMalloc((void **)&d_original, M * N * C * sizeof(uint8_t));
    hipMalloc((void **)&d_inverted, M * N * C * sizeof(uint8_t));

    // Measure total time including data transfer
    hipEvent_t start_cuda_total, stop_cuda_total;
    hipEventCreate(&start_cuda_total);
    hipEventCreate(&stop_cuda_total);
    hipDeviceSynchronize();
    hipEventRecord(start_cuda_total);

    // Copy data to device
    hipMemcpy(d_original, image_array, M * N * C * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Measure kernel execution time separately
    hipEvent_t start_cuda_kernel, stop_cuda_kernel;
    hipEventCreate(&start_cuda_kernel);
    hipEventCreate(&stop_cuda_kernel);
    hipDeviceSynchronize();
    hipEventRecord(start_cuda_kernel);

    // Launch kernel
    REDfilterGPU_Uncoalesced<<<numBlocks, blockSize>>>(d_original, d_inverted, M * N * C);

    // Measure kernel execution time
    hipDeviceSynchronize();
    hipEventRecord(stop_cuda_kernel);
    hipEventSynchronize(stop_cuda_kernel);
    float ms_kernel;
    hipEventElapsedTime(&ms_kernel, start_cuda_kernel, stop_cuda_kernel);

    // Copy result back to host
    hipMemcpy(new_image_array, d_inverted, M * N * C * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // Measure total time including data transfer
    hipDeviceSynchronize();
    hipEventRecord(stop_cuda_total);
    hipEventSynchronize(stop_cuda_total);
    float ms_total;
    hipEventElapsedTime(&ms_total, start_cuda_total, stop_cuda_total);

    // Save the image
    save_image_array(new_image_array);

    // Free memory
    free(new_image_array);
    hipFree(d_original);
    hipFree(d_inverted);

    info info;
    info.kernelTime = ms_kernel / 1000.0; // Convert to seconds
    info.totalTime = ms_total / 1000.0;   // Convert to seconds
    return info;
}



void transformImageData(uint8_t *original, uint8_t *transformed, int size){
    for (int i = 0; i < size / 3; ++i){
        // Place red pixels first
        transformed[i] = original[i * 3]; // Red channel

        // Then green pixels
        transformed[size / 3 + i] = original[i * 3 + 1]; // Green channel

        // Then blue pixels
        transformed[2 * size / 3 + i] = original[i * 3 + 2]; // Blue channel
    }
}

__global__ void redChannelFilter(uint8_t* transformed, uint8_t* inverted, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size / 3) {
        int redIndex = i; // Red pixels are in the first third of the array

        inverted[redIndex] = 255 - transformed[redIndex]; // Inversion of blue channel
    }
}


void reverseTransformImageData(uint8_t *inverted, uint8_t *finalImage, int size){
    for (int i = 0; i < size / 3; ++i){
        // Place the red pixels back to their original location
        finalImage[i * 3] = inverted[i];

        // Place the green pixels back to their original location
        finalImage[i * 3 + 1] = inverted[size / 3 + i];

        // Place the blue pixels back to their original location
        finalImage[i * 3 + 2] = inverted[2 * size / 3 + i];
    }
}

info REDfilterGPUrun_Coalesced() {
    // Read the image
    uint8_t *image_array = get_image_array();
    
    // Allocate host output and transformed input
    uint8_t *new_image_array = (uint8_t *)malloc(M * N * C * sizeof(uint8_t));
    uint8_t *transformed_input = (uint8_t *)malloc(M * N * C * sizeof(uint8_t));

    // Transform the input data
    transformImageData(image_array, transformed_input, M * N * C);

    // Set up grid and block dimensions
    int blockSize = 256;
    int numBlocks = (M * N * C + blockSize - 1) / blockSize;

    // Allocate device memory
    uint8_t *d_transformed, *d_inverted;
    hipMalloc((void **)&d_transformed, M * N * C * sizeof(uint8_t));
    hipMalloc((void **)&d_inverted, M * N * C * sizeof(uint8_t));

    // Measure total time including data transfer
    hipEvent_t start_cuda_total, stop_cuda_total;
    hipEventCreate(&start_cuda_total);
    hipEventCreate(&stop_cuda_total);
    hipDeviceSynchronize();
    hipEventRecord(start_cuda_total);

    // Copy transformed data to device
    hipMemcpy(d_transformed, transformed_input, M * N * C * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Measure kernel execution time separately
    hipEvent_t start_cuda_kernel, stop_cuda_kernel;
    hipEventCreate(&start_cuda_kernel);
    hipEventCreate(&stop_cuda_kernel);
    hipDeviceSynchronize();
    hipEventRecord(start_cuda_kernel);

    // Launch kernels
    redChannelFilter<<<numBlocks, blockSize>>>(d_transformed, d_inverted, M * N * C);
    //greenChannelFilter<<<numBlocks, blockSize>>>(d_transformed, d_inverted, M * N * C);
    //blueChannelFilter<<<numBlocks, blockSize>>>(d_transformed, d_inverted, M * N * C);

    // Measure kernel execution time
    hipDeviceSynchronize();
    hipEventRecord(stop_cuda_kernel);
    hipEventSynchronize(stop_cuda_kernel);
    float ms_kernel;
    hipEventElapsedTime(&ms_kernel, start_cuda_kernel, stop_cuda_kernel);

    // Copy the result back to the host
    hipMemcpy(transformed_input, d_inverted, M * N * C * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // Reverse transform the data
    reverseTransformImageData(transformed_input, new_image_array, M * N * C);

    // Measure total time including data transfer
    hipDeviceSynchronize();
    hipEventRecord(stop_cuda_total);
    hipEventSynchronize(stop_cuda_total);
    float ms_total;
    hipEventElapsedTime(&ms_total, start_cuda_total, stop_cuda_total);

    // Save the image
    save_image_array(new_image_array);

    // Free memory
    free(new_image_array);
    free(transformed_input);
    hipFree(d_transformed);
    hipFree(d_inverted);

    info info;
    info.kernelTime = ms_kernel / 1000.0; // Convert to seconds
    info.totalTime = ms_total / 1000.0;   // Convert to seconds
    return info;
}


float getMedian(std::vector<float> &v){
    size_t n = v.size() / 2;
    std::nth_element(v.begin(), v.begin() + n, v.end());
    return v[n];
}

int main(void){
    const int NUM_RUNS = 20;
    std::vector<float> uncoalesced_times(NUM_RUNS);
    std::vector<float> coalesced_times(NUM_RUNS);

    // Warm-up run
    REDfilterGPUrun_Uncoalesced();
    REDfilterGPUrun_Coalesced();
    hipDeviceSynchronize();

    // Benchmark runs
    for (int i = 0; i < NUM_RUNS; ++i) {
        uncoalesced_times[i] = REDfilterGPUrun_Uncoalesced().kernelTime;
        hipDeviceSynchronize();
        
        coalesced_times[i] = REDfilterGPUrun_Coalesced().kernelTime;
        hipDeviceSynchronize();
    }

    float median_uncoalesced = getMedian(uncoalesced_times);
    float median_coalesced = getMedian(coalesced_times);

    printf("%.6e\n%.6e", median_uncoalesced, median_coalesced);
    return 0;
}