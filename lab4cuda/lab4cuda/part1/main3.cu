/*
 * Code snippet for importing / exporting image data.
 *
 * To convert an image to a pixel map, run `convert <name>.<extension> <name>.ppm
 *
 */
#include <cstdint>  // Data types
#include <iostream> // File operations
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include <iostream>
#include <vector>
#include <random>
#include <ctime>
#include <algorithm>
#include <math.h>

// #define M 512       // Lenna width
// #define N 512       // Lenna height

#define C 3       // Colors
#define OFFSET 16 // Header length


uint8_t *create_random_image_array(int width, int height, int offset) {
    // Seed the random number generator
    srand(time(NULL));

    // Allocate memory for the image array including the offset
    uint8_t *image_array = (uint8_t *)malloc(width * height * 3 * sizeof(uint8_t) + offset);
    if (image_array == NULL) {
        perror("ERROR: Memory allocation failed");
        exit(EXIT_FAILURE);
    }

    // Fill the array with random values (0-255) for RGB components
    for (int i = offset; i < width * height * 3 + offset; i++) {
        image_array[i] = rand() % 256; // Random value between 0 and 255
    }

    // Return the array starting from the offset
    return image_array + offset;
}

void save_image_array(uint8_t *image_array,int M, int N){
    /*
     * Save the data of an (RGB) image as a pixel map.
     *
     * Parameters:
     *  - param1: The data of an (RGB) image as a 1D array
     *
     */
    // Try opening the file
    FILE *imageFile;
    imageFile = fopen("./output_image.ppm", "wb");
    if (imageFile == NULL)
    {
        perror("ERROR: Cannot open output file");
        exit(EXIT_FAILURE);
    }

    // Configure the file
    fprintf(imageFile, "P6\n");          // P6 filetype
    fprintf(imageFile, "%d %d\n", M, N); // dimensions
    fprintf(imageFile, "255\n");         // Max pixel

    // Write the image
    fwrite(image_array, 1, M * N * C, imageFile);

    // Close the file
    fclose(imageFile);
}



struct info{
    float kernelTime;
    float totalTime;
};

__global__ void REDfilterGPU_Uncoalesced(uint8_t *original, uint8_t *inverted, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < size; i += stride) {
        if (i % 3 == 0) {
            inverted[i] = 255 - original[i];
        }
    }
}


info REDfilterGPUrun_Uncoalesced(uint8_t *image_array, int M, int N, int gridSize){ 
    // Read the image
    //uint8_t *image_array = get_image_array();
    // Allocate host output
    uint8_t *new_image_array = (uint8_t *)malloc(M * N * C);

    // Set up grid and block dimensions
    int blockSize = 256;
    int numBlocks = gridSize;

    // Allocate device memory
    uint8_t *d_original, *d_inverted;
    hipMalloc((void **)&d_original, M * N * C * sizeof(uint8_t));
    hipMalloc((void **)&d_inverted, M * N * C * sizeof(uint8_t));

    // Measure total time including data transfer
    hipEvent_t start_cuda_total, stop_cuda_total;
    hipEventCreate(&start_cuda_total);
    hipEventCreate(&stop_cuda_total);
    hipDeviceSynchronize();
    hipEventRecord(start_cuda_total);

    // Copy data to device
    hipMemcpy(d_original, image_array, M * N * C * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Measure kernel execution time separately
    hipEvent_t start_cuda_kernel, stop_cuda_kernel;
    hipEventCreate(&start_cuda_kernel);
    hipEventCreate(&stop_cuda_kernel);
    hipDeviceSynchronize();
    hipEventRecord(start_cuda_kernel);

    // Launch kernel
    REDfilterGPU_Uncoalesced<<<numBlocks, blockSize>>>(d_original, d_inverted, M * N * C);

    // Measure kernel execution time
    hipDeviceSynchronize();
    hipEventRecord(stop_cuda_kernel);
    hipEventSynchronize(stop_cuda_kernel);
    float ms_kernel;
    hipEventElapsedTime(&ms_kernel, start_cuda_kernel, stop_cuda_kernel);

    // Copy result back to host
    hipMemcpy(new_image_array, d_inverted, M * N * C * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // Measure total time including data transfer
    hipDeviceSynchronize();
    hipEventRecord(stop_cuda_total);
    hipEventSynchronize(stop_cuda_total);
    float ms_total;
    hipEventElapsedTime(&ms_total, start_cuda_total, stop_cuda_total);

    // Save the image
    save_image_array(new_image_array,M,N);

    // Free memory
    free(new_image_array);
    hipFree(d_original);
    hipFree(d_inverted);

    info info;
    info.kernelTime = ms_kernel / 1000.0; // Convert to seconds
    info.totalTime = ms_total / 1000.0;   // Convert to seconds
    return info;
}



void transformImageData(uint8_t *original, uint8_t *transformed, int size){
    for (int i = 0; i < size / 3; ++i){
        // Place red pixels first
        transformed[i] = original[i * 3]; // Red channel

        // Then green pixels
        transformed[size / 3 + i] = original[i * 3 + 1]; // Green channel

        // Then blue pixels
        transformed[2 * size / 3 + i] = original[i * 3 + 2]; // Blue channel
    }
}

__global__ void redChannelFilter(uint8_t* transformed, uint8_t* inverted, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < size / 3; i += stride) {
        inverted[i] = 255 - transformed[i];
    }
}


void reverseTransformImageData(uint8_t *inverted, uint8_t *finalImage, int size){
    for (int i = 0; i < size / 3; ++i){
        // Place the red pixels back to their original location
        finalImage[i * 3] = inverted[i];

        // Place the green pixels back to their original location
        finalImage[i * 3 + 1] = inverted[size / 3 + i];

        // Place the blue pixels back to their original location
        finalImage[i * 3 + 2] = inverted[2 * size / 3 + i];
    }
}

info REDfilterGPUrun_Coalesced(uint8_t *image_array, int M, int N, int gridSize) {
    // Read the image
    //uint8_t *image_array = get_image_array();
    
    
    // Allocate host output and transformed input
    uint8_t *new_image_array = (uint8_t *)malloc(M * N * C * sizeof(uint8_t));
    uint8_t *transformed_input = (uint8_t *)malloc(M * N * C * sizeof(uint8_t));

    // Transform the input data
    transformImageData(image_array, transformed_input, M * N * C);

    // Set up grid and block dimensions
    int blockSize = 256;
    int numBlocks = gridSize;

    // Allocate device memory
    uint8_t *d_transformed, *d_inverted;
    hipMalloc((void **)&d_transformed, M * N * C * sizeof(uint8_t));
    hipMalloc((void **)&d_inverted, M * N * C * sizeof(uint8_t));

    // Measure total time including data transfer
    hipEvent_t start_cuda_total, stop_cuda_total;
    hipEventCreate(&start_cuda_total);
    hipEventCreate(&stop_cuda_total);
    hipDeviceSynchronize();
    hipEventRecord(start_cuda_total);

    // Copy transformed data to device
    hipMemcpy(d_transformed, transformed_input, M * N * C * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Measure kernel execution time separately
    hipEvent_t start_cuda_kernel, stop_cuda_kernel;
    hipEventCreate(&start_cuda_kernel);
    hipEventCreate(&stop_cuda_kernel);
    hipDeviceSynchronize();
    hipEventRecord(start_cuda_kernel);

    // Launch kernels
    redChannelFilter<<<numBlocks, blockSize>>>(d_transformed, d_inverted, M * N * C);
    //greenChannelFilter<<<numBlocks, blockSize>>>(d_transformed, d_inverted, M * N * C);
    //blueChannelFilter<<<numBlocks, blockSize>>>(d_transformed, d_inverted, M * N * C);

    // Measure kernel execution time
    hipDeviceSynchronize();
    hipEventRecord(stop_cuda_kernel);
    hipEventSynchronize(stop_cuda_kernel);
    float ms_kernel;
    hipEventElapsedTime(&ms_kernel, start_cuda_kernel, stop_cuda_kernel);

    // Copy the result back to the host
    hipMemcpy(transformed_input, d_inverted, M * N * C * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // Reverse transform the data
    reverseTransformImageData(transformed_input, new_image_array, M * N * C);

    // Measure total time including data transfer
    hipDeviceSynchronize();
    hipEventRecord(stop_cuda_total);
    hipEventSynchronize(stop_cuda_total);
    float ms_total;
    hipEventElapsedTime(&ms_total, start_cuda_total, stop_cuda_total);

    // Save the image
    save_image_array(new_image_array,M,N);

    // Free memory
    free(new_image_array);
    free(transformed_input);
    hipFree(d_transformed);
    hipFree(d_inverted);

    info info;
    info.kernelTime = ms_kernel / 1000.0; // Convert to seconds
    info.totalTime = ms_total / 1000.0;   // Convert to seconds
    return info;
}


float getMedian(std::vector<float> &v){
    size_t n = v.size() / 2;
    std::nth_element(v.begin(), v.begin() + n, v.end());
    return v[n];
}


int main(void){
    const int NUM_RUNS = 20;
    const int num_grid_sizes = 7;
    const int grid_sizes[num_grid_sizes] = {16, 32, 64, 128, 256,512,1024};

    int height = 512;
    int width = 512;

    
    //note that we do implement striding in this code
    for(int i = 0; i < num_grid_sizes; i++){
        int gridSize = grid_sizes[i];
        std::vector<float> uncoalesced_times(NUM_RUNS);
        std::vector<float> coalesced_times(NUM_RUNS);

        for (int j = 0; j < NUM_RUNS; j++) {
            // Create a random image array
            uint8_t *image_array_uncoalesced = create_random_image_array(width, height, OFFSET);
            uint8_t *image_array_coalesced = create_random_image_array(width, height, OFFSET);
            

            // Define M and N here, since they are used in the kernel calls
            int M = width;
            int N = height;

            // Run the uncoalesced kernel
            info uncoalesced_info = REDfilterGPUrun_Uncoalesced(image_array_uncoalesced, M, N, gridSize);
            uncoalesced_times[j] = uncoalesced_info.kernelTime;

            // Run the coalesced kernel
            info coalesced_info = REDfilterGPUrun_Coalesced(image_array_coalesced, M, N, gridSize);
            coalesced_times[j] = coalesced_info.kernelTime;

            // Free the image array
            free(image_array_uncoalesced - OFFSET);
            free(image_array_coalesced - OFFSET);

        }
        //print the median of the uncoalesced and coalesced times
        float median_uncoalesced = getMedian(uncoalesced_times);    
        float median_coalesced = getMedian(coalesced_times);

        printf(" %d,%f,%f\n", gridSize, median_uncoalesced, median_coalesced);
    }

    return 0;
}