/*
 * Code snippet for importing / exporting image data.
 *
 * To convert an image to a pixel map, run `convert <name>.<extension> <name>.ppm
 *
 */
#include <cstdint>  // Data types
#include <iostream> // File operations
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include <iostream>
#include <vector>
#include <random>
#include <ctime>
#include <algorithm>
#include <math.h>

// #define M 512       // Lenna width
// #define N 512       // Lenna height
#define M 960     // VR width
#define N 1280    // VR height
#define C 3       // Colors
#define OFFSET 16 // Header length

uint8_t *get_image_array(void)
{
    /*
     * Get the data of an (RGB) image as a 1D array.
     *
     * Returns: Flattened image array.
     *
     * Noets:
     *  - Images data is flattened per color, column, row.
     *  - The first 3 data elements are the RGB components
     *  - The first 3*M data elements represent the firts row of the image
     *  - For example, r_{0,0}, g_{0,0}, b_{0,0}, ..., b_{0,M}, r_{1,0}, ..., b_{b,M}, ..., b_{N,M}
     *
     */
    // Try opening the file
    FILE *imageFile;
    imageFile = fopen("./anna.ppm", "rb");
    if (imageFile == NULL)
    {
        perror("ERROR: Cannot open output file");
        exit(EXIT_FAILURE);
    }

    // Initialize empty image array
    uint8_t *image_array = (uint8_t *)malloc(M * N * C * sizeof(uint8_t) + OFFSET);

    // Read the image
    fread(image_array, sizeof(uint8_t), M * N * C * sizeof(uint8_t) + OFFSET, imageFile);

    // Close the file
    fclose(imageFile);

    // Move the starting pointer and return the flattened image array
    return image_array + OFFSET;
}

void save_image_array(uint8_t *image_array)
{
    /*
     * Save the data of an (RGB) image as a pixel map.
     *
     * Parameters:
     *  - param1: The data of an (RGB) image as a 1D array
     *
     */
    // Try opening the file
    FILE *imageFile;
    imageFile = fopen("./output_image.ppm", "wb");
    if (imageFile == NULL)
    {
        perror("ERROR: Cannot open output file");
        exit(EXIT_FAILURE);
    }

    // Configure the file
    fprintf(imageFile, "P6\n");          // P6 filetype
    fprintf(imageFile, "%d %d\n", M, N); // dimensions
    fprintf(imageFile, "255\n");         // Max pixel

    // Write the image
    fwrite(image_array, 1, M * N * C, imageFile);

    // Close the file
    fclose(imageFile);
}

struct info
{
    float kernelTime;
    float totalTime;
};

uint8_t *inversionCPU(uint8_t *original, uint8_t *inverted)
{
    for (int i = 0; i < M * N * C; i += 3)
    {
        inverted[i] = 255 - original[i];
    }
    for (int i = 1; i < M * N * C; i += 3)
    {
        inverted[i] = 255 - original[i];
    }
    for (int i = 2; i < M * N * C; i += 3)
    {
        inverted[i] = 255 - original[i];
    }
    return inverted;
}

uint8_t *REDfilterCPU(uint8_t *original, uint8_t *inverted)
{
    for (int i = 0; i < M * N * C; i += 3)
    {
        inverted[i] = 0.1 * original[i - 2] + 0.25 * original[i - 1] + 0.5 * original[i] + 0.25 * original[i + 1] + 0.1 * original[i + 2];
    }
    for (int i = 1; i < M * N * C; i += 3)
    {
        inverted[i] = 255 - original[i];
    }
    for (int i = 2; i < M * N * C; i += 3)
    {
        inverted[i] = 255 - original[i];
    }
    return inverted;
}

__global__ void inversionGPU(uint8_t *original, uint8_t *inverted, int size)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x)
    {
        inverted[i] = 255 - original[i];
    }
}

__global__ void REDfilterGPU_WithThreadDivergence(uint8_t *original, uint8_t *inverted, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size)
    {
        if (i % 3 != 0)
        {
            inverted[i] = 255 - original[i];
        }
        else
        {
            inverted[i] = 0.1 * original[i - 2] + 0.25 * original[i - 1] + 0.5 * original[i] + 0.25 * original[i + 1] + 0.1 * original[i + 2];
        }
    }
}



info inversionCPUrun()
{
    // Read the image
    uint8_t *image_array = get_image_array();
    // Allocate host output
    uint8_t *new_image_array = (uint8_t *)malloc(M * N * C);
    // CPU
    // measuring
    const auto start = std::chrono::steady_clock::now();
    // measuring

    inversionCPU(image_array, new_image_array);

    // measuring
    const auto end = std::chrono::steady_clock::now();
    const std::chrono::duration<double> elapsed_seconds{end - start};
    // measuring

    info info;
    info.kernelTime = elapsed_seconds.count(); // Convert to seconds
    info.totalTime = elapsed_seconds.count();  // Convert to seconds

    return info;
}

info REDfilterCPUrun()
{
    // Read the image
    uint8_t *image_array = get_image_array();
    // Allocate host output
    uint8_t *new_image_array = (uint8_t *)malloc(M * N * C);
    // CPU
    // measuring
    const auto start = std::chrono::steady_clock::now();
    // measuring

    REDfilterCPU(image_array, new_image_array);

    // measuring
    const auto end = std::chrono::steady_clock::now();
    const std::chrono::duration<double> elapsed_seconds{end - start};
    // measuring

    info info;
    info.kernelTime = elapsed_seconds.count(); // Convert to seconds
    info.totalTime = elapsed_seconds.count();  // Convert to seconds

    return info;
}

info inversionGPUrun(int input_blockSize)
{
    // Read the image
    uint8_t *image_array = get_image_array();
    // Allocate host output
    uint8_t *new_image_array = (uint8_t *)malloc(M * N * C);

    // Set up grid and block dimensions
    int blockSize = input_blockSize;
    int numBlocks = (M * N * C + blockSize - 1) / blockSize / 10;

    // Allocate device memory
    uint8_t *d_original, *d_inverted;
    hipMalloc((void **)&d_original, M * N * C * sizeof(uint8_t));
    hipMalloc((void **)&d_inverted, M * N * C * sizeof(uint8_t));

    // Measure total time including data transfer
    hipEvent_t start_cuda_total, stop_cuda_total;
    hipEventCreate(&start_cuda_total);
    hipEventCreate(&stop_cuda_total);
    hipDeviceSynchronize();
    hipEventRecord(start_cuda_total);

    // Copy data to device
    hipMemcpy(d_original, image_array, M * N * C * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Measure kernel execution time separately
    hipEvent_t start_cuda_kernel, stop_cuda_kernel;
    hipEventCreate(&start_cuda_kernel);
    hipEventCreate(&stop_cuda_kernel);
    hipDeviceSynchronize();
    hipEventRecord(start_cuda_kernel);

    // Launch kernel
    inversionGPU<<<numBlocks, blockSize>>>(d_original, d_inverted, M * N * C);

    // Measure kernel execution time
    hipDeviceSynchronize();
    hipEventRecord(stop_cuda_kernel);
    hipEventSynchronize(stop_cuda_kernel);
    float ms_kernel;
    hipEventElapsedTime(&ms_kernel, start_cuda_kernel, stop_cuda_kernel);

    // Copy result back to host
    hipMemcpy(new_image_array, d_inverted, M * N * C * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // Measure total time including data transfer
    hipDeviceSynchronize();
    hipEventRecord(stop_cuda_total);
    hipEventSynchronize(stop_cuda_total);
    float ms_total;
    hipEventElapsedTime(&ms_total, start_cuda_total, stop_cuda_total);

    // Save the image
    save_image_array(new_image_array);

    // Free memory
    free(new_image_array);
    hipFree(d_original);
    hipFree(d_inverted);

    info info;
    info.kernelTime = ms_kernel / 1000.0; // Convert to seconds
    info.totalTime = ms_total / 1000.0;   // Convert to seconds
    return info;
}

info REDfilterGPUrun_WithThreadDivergence()
{
    // Read the image
    uint8_t *image_array = get_image_array();
    // Allocate host output
    uint8_t *new_image_array = (uint8_t *)malloc(M * N * C);

    // Set up grid and block dimensions
    int blockSize = 256;
    int numBlocks = (M * N * C + blockSize - 1) / blockSize;

    // Allocate device memory
    uint8_t *d_original, *d_inverted;
    hipMalloc((void **)&d_original, M * N * C * sizeof(uint8_t));
    hipMalloc((void **)&d_inverted, M * N * C * sizeof(uint8_t));

    // Measure total time including data transfer
    hipEvent_t start_cuda_total, stop_cuda_total;
    hipEventCreate(&start_cuda_total);
    hipEventCreate(&stop_cuda_total);
    hipDeviceSynchronize();
    hipEventRecord(start_cuda_total);

    // Copy data to device
    hipMemcpy(d_original, image_array, M * N * C * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Measure kernel execution time separately
    hipEvent_t start_cuda_kernel, stop_cuda_kernel;
    hipEventCreate(&start_cuda_kernel);
    hipEventCreate(&stop_cuda_kernel);
    hipDeviceSynchronize();
    hipEventRecord(start_cuda_kernel);

    // Launch kernel
    REDfilterGPU_WithThreadDivergence<<<numBlocks, blockSize>>>(d_original, d_inverted, M * N * C);

    // Measure kernel execution time
    hipDeviceSynchronize();
    hipEventRecord(stop_cuda_kernel);
    hipEventSynchronize(stop_cuda_kernel);
    float ms_kernel;
    hipEventElapsedTime(&ms_kernel, start_cuda_kernel, stop_cuda_kernel);

    // Copy result back to host
    hipMemcpy(new_image_array, d_inverted, M * N * C * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // Measure total time including data transfer
    hipDeviceSynchronize();
    hipEventRecord(stop_cuda_total);
    hipEventSynchronize(stop_cuda_total);
    float ms_total;
    hipEventElapsedTime(&ms_total, start_cuda_total, stop_cuda_total);

    // Save the image
    save_image_array(new_image_array);

    // Free memory
    free(new_image_array);
    hipFree(d_original);
    hipFree(d_inverted);

    info info;
    info.kernelTime = ms_kernel / 1000.0; // Convert to seconds
    info.totalTime = ms_total / 1000.0;   // Convert to seconds
    return info;
}

float getMedian(std::vector<float> &v)
{
    size_t n = v.size() / 2;
    std::nth_element(v.begin(), v.begin() + n, v.end());
    return v[n];
}

/* int main (void) {
    const int NUM_RUNS = 80;

    // Read the image
    //uint8_t* image_array = get_image_array();

    // Allocate output
    //uint8_t* new_image_array = (uint8_t*)malloc(M*N*C);

    std::vector<float> timesInversionCPU(NUM_RUNS);
    std::vector<float> timesInversionGPU(NUM_RUNS);
    std::vector<float> timesREDfilterCPU(NUM_RUNS);
    std::vector<float> timesREDfilterGPU(NUM_RUNS);

    for (int run = 0; run < NUM_RUNS; run++) {
        cudaDeviceSynchronize();
        timesInversionCPU[run] = inversionCPUrun().kernelTime;
        cudaDeviceSynchronize();
        timesInversionGPU[run] = inversionGPUrun().kernelTime;
        cudaDeviceSynchronize();
        timesREDfilterCPU[run] = REDfilterCPUrun().kernelTime;
        cudaDeviceSynchronize();
        timesREDfilterGPU[run] = REDfilterGPUrun().kernelTime;
        cudaDeviceSynchronize();
    }

    float medianInversionCPU = getMedian(timesInversionCPU);
    float medianInversionGPU = getMedian(timesInversionGPU);
    float medianREDfilterCPU = getMedian(timesREDfilterCPU);
    float medianREDfilterGPU = getMedian(timesREDfilterGPU);

    printf("%.6e\n", medianInversionCPU);
    printf("%.6e\n", medianInversionGPU);
    printf("%.6e\n", medianREDfilterCPU);
    printf("%.6e\n", medianREDfilterGPU);

    //Free allocated memory
    //free(new_image_array);
    //free(image_array);

    return 0;
} */

/* int main(void) {
    const int NUM_RUNS = 10;
    const int MAX_blocksize = 380;


    for (int blocksize = 1; blocksize < MAX_blocksize; blocksize++) {
        std::vector<float> timesInversionGPU(NUM_RUNS);

        for (int run = 0; run < NUM_RUNS; run++) {
            cudaDeviceSynchronize();
            timesInversionGPU[run] = inversionGPUrun(blocksize).kernelTime;
            cudaDeviceSynchronize();
        }

        float medianInversionGPU = getMedian(timesInversionGPU);
        if(blocksize < MAX_blocksize-3){
            printf("%d, %.6e\n", blocksize, medianInversionGPU);
        }
    }
    return 0;
} */

void transformImageData(uint8_t *original, uint8_t *transformed, int size)
{
    for (int i = 0; i < size / 3; ++i)
    {
        // Place red pixels first
        transformed[i] = original[i * 3]; // Red channel

        // Then green pixels
        transformed[size / 3 + i] = original[i * 3 + 1]; // Green channel

        // Then blue pixels
        transformed[2 * size / 3 + i] = original[i * 3 + 2]; // Blue channel
    }
}

__global__ void redChannelFilter(uint8_t* transformed, uint8_t* inverted, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size / 3) {
        int redIndex = i; // Red pixels are in the first third of the array

        // Avoid out-of-bounds memory access by checking the index range
        if (i > 1 && i < size / 3 - 2) {
            inverted[redIndex] = 0.1 * transformed[redIndex - 2] + 0.25 * transformed[redIndex - 1] + 0.5 * transformed[redIndex] + 0.25 * transformed[redIndex + 1] + 0.1 * transformed[redIndex + 2];
        } else {
            inverted[redIndex] = transformed[redIndex]; // No filtering for boundary pixels
        }
    }
}

__global__ void blueChannelInversion(uint8_t* transformed, uint8_t* inverted, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size / 3) {
        int blueIndex = i + 2 * size / 3; // Blue pixels are in the last third of the array
        inverted[blueIndex] = 255 - transformed[blueIndex]; // Inversion of blue channel
    }
}

__global__ void greenChannelInversion(uint8_t* transformed, uint8_t* inverted, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size / 3) {
        int greenIndex = i + size / 3; // Green pixels are in the second third of the array
        inverted[greenIndex] = 255 - transformed[greenIndex]; // Inversion of green channel
    }
}

void reverseTransformImageData(uint8_t *inverted, uint8_t *finalImage, int size)
{
    for (int i = 0; i < size / 3; ++i)
    {
        // Place the red pixels back to their original location
        finalImage[i * 3] = inverted[i];

        // Place the green pixels back to their original location
        finalImage[i * 3 + 1] = inverted[size / 3 + i];

        // Place the blue pixels back to their original location
        finalImage[i * 3 + 2] = inverted[2 * size / 3 + i];
    }
}




info REDfilterGPUrun_WithoutThreadDivergence() {
    // Read the image
    uint8_t *image_array = get_image_array();
    
    // Allocate host output and transformed input
    uint8_t *new_image_array = (uint8_t *)malloc(M * N * C * sizeof(uint8_t));
    uint8_t *transformed_input = (uint8_t *)malloc(M * N * C * sizeof(uint8_t));

    // Transform the input data
    transformImageData(image_array, transformed_input, M * N * C);

    // Set up grid and block dimensions
    int blockSize = 256;
    int numBlocks = (M * N * C + blockSize - 1) / blockSize;

    // Allocate device memory
    uint8_t *d_transformed, *d_inverted;
    hipMalloc((void **)&d_transformed, M * N * C * sizeof(uint8_t));
    hipMalloc((void **)&d_inverted, M * N * C * sizeof(uint8_t));

    // Measure total time including data transfer
    hipEvent_t start_cuda_total, stop_cuda_total;
    hipEventCreate(&start_cuda_total);
    hipEventCreate(&stop_cuda_total);
    hipDeviceSynchronize();
    hipEventRecord(start_cuda_total);

    // Copy transformed data to device
    hipMemcpy(d_transformed, transformed_input, M * N * C * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Measure kernel execution time separately
    hipEvent_t start_cuda_kernel, stop_cuda_kernel;
    hipEventCreate(&start_cuda_kernel);
    hipEventCreate(&stop_cuda_kernel);
    hipDeviceSynchronize();
    hipEventRecord(start_cuda_kernel);

    // Launch kernels
    redChannelFilter<<<numBlocks, blockSize>>>(d_transformed, d_inverted, M * N * C);
    greenChannelInversion<<<numBlocks, blockSize>>>(d_transformed, d_inverted, M * N * C);
    blueChannelInversion<<<numBlocks, blockSize>>>(d_transformed, d_inverted, M * N * C);

    // Measure kernel execution time
    hipDeviceSynchronize();
    hipEventRecord(stop_cuda_kernel);
    hipEventSynchronize(stop_cuda_kernel);
    float ms_kernel;
    hipEventElapsedTime(&ms_kernel, start_cuda_kernel, stop_cuda_kernel);

    // Copy the result back to the host
    hipMemcpy(transformed_input, d_inverted, M * N * C * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // Reverse transform the data
    reverseTransformImageData(transformed_input, new_image_array, M * N * C);

    // Measure total time including data transfer
    hipDeviceSynchronize();
    hipEventRecord(stop_cuda_total);
    hipEventSynchronize(stop_cuda_total);
    float ms_total;
    hipEventElapsedTime(&ms_total, start_cuda_total, stop_cuda_total);

    // Save the image
    save_image_array(new_image_array);

    // Free memory
    free(new_image_array);
    free(transformed_input);
    hipFree(d_transformed);
    hipFree(d_inverted);

    info info;
    info.kernelTime = ms_kernel / 1000.0; // Convert to seconds
    info.totalTime = ms_total / 1000.0;   // Convert to seconds
    return info;
}

int main(void)
{
    float runZero = REDfilterGPUrun_WithThreadDivergence().kernelTime; // Warm up
    hipDeviceSynchronize();
    float runOne = REDfilterGPUrun_WithThreadDivergence().kernelTime;
    hipDeviceSynchronize();

    float runTwo = REDfilterGPUrun_WithoutThreadDivergence().kernelTime; // Warm up
    hipDeviceSynchronize();
    float runThree = REDfilterGPUrun_WithoutThreadDivergence().kernelTime;
    hipDeviceSynchronize();

    printf("%.6e\n%.6e", runOne, runThree);
    return 0;
}