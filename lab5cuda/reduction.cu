#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include <iostream>
#include <vector>
#include <random>
#include <ctime>
#include <algorithm>
#include <math.h>
#include <cfloat>

struct info {
    float kernelTime;
    float totalTime;
    float outcome[4];
};

__global__ void reductionMAXGPU(float *d_array, int size) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    sdata[tid] = (i < size) ? d_array[i] : -FLT_MAX;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = max(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }
    if (tid == 0) d_array[blockIdx.x] = sdata[0];
}


__global__ void reductionMINGPU(float *d_array, int size) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = (i < size) ? d_array[i] : FLT_MAX;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = min(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }
    if (tid == 0) d_array[blockIdx.x] = sdata[0];
}

__global__ void reductionPRODGPU(float *d_array, int size) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    sdata[tid] = (i < size) ? d_array[i] : 1;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = sdata[tid] * sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) d_array[blockIdx.x] = sdata[0];
}

__global__ void reductionSUMGPU(float *d_array, int size) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    sdata[tid] = (i < size) ? d_array[i] : 0;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = sdata[tid] + sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) d_array[blockIdx.x] = sdata[0];
}

info runGPUReductionSync(float array_SUM[], float array_PROD[], float array_MIN[], float array_MAX[],float size){
    float *GPU_d_array_SUM;
    float *GPU_d_array_PROD;
    float *GPU_d_array_MIN;
    float *GPU_d_array_MAX;

    hipMalloc((void**)&GPU_d_array_SUM, size * sizeof(float));
    hipMalloc((void**)&GPU_d_array_PROD, size * sizeof(float));
    hipMalloc((void**)&GPU_d_array_MIN, size * sizeof(float));
    hipMalloc((void**)&GPU_d_array_MAX, size * sizeof(float));

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    int sharedMemSize = threadsPerBlock * sizeof(float);

    // Measure total time including data transfer
    hipEvent_t start_cuda_total, stop_cuda_total;
    hipEventCreate(&start_cuda_total);
    hipEventCreate(&stop_cuda_total);
    hipDeviceSynchronize();
    hipEventRecord(start_cuda_total);

    // Measure kernel execution time separately
    hipEvent_t start_cuda_kernel, stop_cuda_kernel;
    hipEventCreate(&start_cuda_kernel);
    hipEventCreate(&stop_cuda_kernel);
    hipDeviceSynchronize();
    hipEventRecord(start_cuda_kernel);

    hipMemcpy(GPU_d_array_SUM, array_SUM, size * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    reductionSUMGPU<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(GPU_d_array_SUM, size);
    hipDeviceSynchronize();
    hipMemcpy(GPU_d_array_PROD, array_PROD, size * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    reductionPRODGPU<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(GPU_d_array_PROD, size);
    hipDeviceSynchronize();
    hipMemcpy(GPU_d_array_MIN, array_MIN, size * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    reductionMINGPU<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(GPU_d_array_MIN, size);
    hipDeviceSynchronize();
    hipMemcpy(GPU_d_array_MAX, array_MAX, size * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    reductionMAXGPU<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(GPU_d_array_MAX, size);
    hipDeviceSynchronize();
    

    // Measure kernel execution time
    hipDeviceSynchronize();
    hipEventRecord(stop_cuda_kernel);
    hipEventSynchronize(stop_cuda_kernel);
    float ms_kernel;
    hipEventElapsedTime(&ms_kernel, start_cuda_kernel, stop_cuda_kernel);

    hipMemcpy(array_SUM, GPU_d_array_SUM, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(array_PROD, GPU_d_array_PROD, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(array_MIN, GPU_d_array_MIN, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(array_MAX, GPU_d_array_MAX, sizeof(float), hipMemcpyDeviceToHost);

    // Measure total time including data transfer
    hipDeviceSynchronize();
    hipEventRecord(stop_cuda_total);
    hipEventSynchronize(stop_cuda_total);
    float ms_total;
    hipEventElapsedTime(&ms_total, start_cuda_total, stop_cuda_total);

    hipFree(GPU_d_array_SUM);
    hipFree(GPU_d_array_PROD);
    hipFree(GPU_d_array_MIN);
    hipFree(GPU_d_array_MAX);

    info info;
    info.kernelTime = ms_kernel / 1000.0; 
    info.totalTime = ms_total / 1000.0;   
    info.outcome[0] = array_SUM[0];
    info.outcome[1] =  array_PROD[0];
    info.outcome[2] = array_MIN[0];
    info.outcome[3] =  array_MAX[0];
    return info;
}

info runGPUReductionAsync(float array_SUM[], float array_PROD[], float array_MIN[], float array_MAX[], float size) {
    float *GPU_d_array_SUM, *GPU_d_array_PROD, *GPU_d_array_MIN, *GPU_d_array_MAX;
    hipStream_t stream1;
    hipStream_t stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    hipMalloc((void**)&GPU_d_array_SUM, size * sizeof(float));
    hipMalloc((void**)&GPU_d_array_PROD, size * sizeof(float));
    hipMalloc((void**)&GPU_d_array_MIN, size * sizeof(float));
    hipMalloc((void**)&GPU_d_array_MAX, size * sizeof(float));

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    int sharedMemSize = threadsPerBlock * sizeof(float);

    hipEvent_t start_cuda_total, stop_cuda_total, start_cuda_kernel, stop_cuda_kernel;
    hipEventCreate(&start_cuda_total);
    hipEventCreate(&stop_cuda_total);
    hipEventCreate(&start_cuda_kernel);
    hipEventCreate(&stop_cuda_kernel);

    hipEventRecord(start_cuda_total, stream1);
    hipEventRecord(start_cuda_kernel, stream1);

    // Asynchronous memory copies and kernel launches
    hipMemcpyAsync(GPU_d_array_SUM, array_SUM, size * sizeof(float), hipMemcpyHostToDevice, stream1);
    reductionSUMGPU<<<blocksPerGrid, threadsPerBlock, sharedMemSize, stream2>>>(GPU_d_array_SUM, size);

    hipMemcpyAsync(GPU_d_array_PROD, array_PROD, size * sizeof(float), hipMemcpyHostToDevice, stream1);
    reductionPRODGPU<<<blocksPerGrid, threadsPerBlock, sharedMemSize, stream2>>>(GPU_d_array_PROD, size);

    hipMemcpyAsync(GPU_d_array_MIN, array_MIN, size * sizeof(float), hipMemcpyHostToDevice, stream1);
    reductionMINGPU<<<blocksPerGrid, threadsPerBlock, sharedMemSize, stream2>>>(GPU_d_array_MIN, size);

    hipMemcpyAsync(GPU_d_array_MAX, array_MAX, size * sizeof(float), hipMemcpyHostToDevice, stream1);
    reductionMAXGPU<<<blocksPerGrid, threadsPerBlock, sharedMemSize, stream2>>>(GPU_d_array_MAX, size);

    hipEventRecord(stop_cuda_kernel, stream1);

    // Asynchronous copy of results back to host
    hipMemcpyAsync(array_SUM, GPU_d_array_SUM, sizeof(float), hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(array_PROD, GPU_d_array_PROD, sizeof(float), hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(array_MIN, GPU_d_array_MIN, sizeof(float), hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(array_MAX, GPU_d_array_MAX, sizeof(float), hipMemcpyDeviceToHost, stream1);

    hipEventRecord(stop_cuda_total, stream1);
    hipStreamSynchronize(stream1);

    float ms_kernel, ms_total;
    hipEventElapsedTime(&ms_kernel, start_cuda_kernel, stop_cuda_kernel);
    hipEventElapsedTime(&ms_total, start_cuda_total, stop_cuda_total);

    hipFree(GPU_d_array_SUM);
    hipFree(GPU_d_array_PROD);
    hipFree(GPU_d_array_MIN);
    hipFree(GPU_d_array_MAX);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    info info;
    info.kernelTime = ms_kernel / 1000.0;
    info.totalTime = ms_total / 1000.0;
    info.outcome[0] = array_SUM[0];
    info.outcome[1] = array_PROD[0];
    info.outcome[2] = array_MIN[0];
    info.outcome[3] = array_MAX[0];
    return info;
}

float* generateRandomArray(size_t size) {
    float* array = new float[size];
    
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(0.0f, 1.0f); 

    for (size_t i = 0; i < size; ++i) {
        do {
            array[i] = dis(gen);
        } while (array[i] == 0.0f || array[i] == 1.0f);
    }

    std::uniform_int_distribution<size_t> posDis(0, size - 1);
    size_t maxPosition = posDis(gen);

    array[maxPosition] = 0.999999f;
    return array;
}

void printArray(int* array, size_t size) {
    for (size_t i = 0; i < size; ++i) {
        std::cout << array[i] << " ";
    }
    std::cout << std::endl;
    printf("\n");
}

float getMedian(std::vector<float>& v) {
    size_t n = v.size() / 2;
    std::nth_element(v.begin(), v.begin()+n, v.end());
    return v[n];
}

int main() {
    const int NUM_RUNS = 20;
    const int NUM_SIZES = 11;
    const int sizes[NUM_SIZES] = {512, 1024, 2048, 4096, 8192, 16384, 32768, 65536, 131072, 262144, 524288};

    for (int size_index = 0; size_index < NUM_SIZES; size_index++) {
        int size = sizes[size_index];
        std::vector<float> sync_kernel_times(NUM_RUNS);
        std::vector<float> sync_total_times(NUM_RUNS);
        std::vector<float> async_kernel_times(NUM_RUNS);
        std::vector<float> async_total_times(NUM_RUNS);

        for (int run = 0; run < NUM_RUNS; run++) {
            float* temp = generateRandomArray(size);
            float* my_array = new float[size];
            memcpy(my_array, temp, size * sizeof(float));
            delete[] temp;

            // Synchronous run
            float* my_array_SUM = new float[size];
            float* my_array_PROD = new float[size];
            float* my_array_MIN = new float[size];
            float* my_array_MAX = new float[size];

            memcpy(my_array_SUM, my_array, size * sizeof(float));
            memcpy(my_array_PROD, my_array, size * sizeof(float));
            memcpy(my_array_MIN, my_array, size * sizeof(float));
            memcpy(my_array_MAX, my_array, size * sizeof(float));

            info results_sync = runGPUReductionSync(my_array_SUM, my_array_PROD, my_array_MIN, my_array_MAX, size);
            sync_kernel_times[run] = results_sync.kernelTime;
            sync_total_times[run] = results_sync.totalTime;

            // Asynchronous run
            float* my_array_SUM2 = new float[size];
            float* my_array_PROD2 = new float[size];
            float* my_array_MIN2 = new float[size];
            float* my_array_MAX2 = new float[size];

            memcpy(my_array_SUM2, my_array, size * sizeof(float));
            memcpy(my_array_PROD2, my_array, size * sizeof(float));
            memcpy(my_array_MIN2, my_array, size * sizeof(float));
            memcpy(my_array_MAX2, my_array, size * sizeof(float));

            info results_async = runGPUReductionAsync(my_array_SUM2, my_array_PROD2, my_array_MIN2, my_array_MAX2, size);
            async_kernel_times[run] = results_async.kernelTime;
            async_total_times[run] = results_async.totalTime;

            delete[] my_array;
            delete[] my_array_SUM;
            delete[] my_array_PROD;
            delete[] my_array_MIN;
            delete[] my_array_MAX;
            delete[] my_array_SUM2;
            delete[] my_array_PROD2;
            delete[] my_array_MIN2;
            delete[] my_array_MAX2;
        }
        float sync_total_median = getMedian(sync_total_times);
        float async_total_median = getMedian(async_total_times);
        printf("%d,%.6e,%.6e\n", size, sync_total_median, async_total_median);
    }
    return 0;
}